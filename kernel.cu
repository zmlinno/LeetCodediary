﻿
#include "hip/hip_runtime.h"

#include<hipsparse.h>
#include<stdio.h>
#include<iostream>
#include<chrono>
#include<algorithm>
#include<vector>
#include<device_atomic_functions.h>
#include<hip/device_functions.h>

#include<sstream>
#include<fstream>
using namespace std;
//今天也不能忘了
// 错误检查宏
// 错误检查宏
#define CUDA_CHECK(err) \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl; \
        exit(EXIT_FAILURE); \
    }

#define CUSPARSE_CHECK(err) \
    if (err != HIPSPARSE_STATUS_SUCCESS) { \
        std::cerr << "cuSPARSE Error: " << err << std::endl; \
        exit(EXIT_FAILURE); \
    }

// 解析 Matrix Market 格式文件
void readMatrixMarket(const std::string & filename, int& rows, int& cols, int& nnz,
    std::vector<int>&row_indices, std::vector<int>&col_indices, std::vector<float>&values) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Unable to open file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    std::string line;
    while (std::getline(file, line)) {
        if (line[0] != '%') break; // 跳过注释行
    }

    std::istringstream iss(line);
    iss >> rows >> cols >> nnz;

    if (rows <= 0 || cols <= 0 || nnz <= 0) {
        std::cerr << "Error: Invalid matrix dimensions or non-zero count." << std::endl;
        exit(EXIT_FAILURE);
    }

    row_indices.reserve(nnz);
    col_indices.reserve(nnz);
    values.reserve(nnz);

    int row, col;
    float value;
    while (file >> row >> col >> value) {
        if (row <= 0 || col <= 0) {
            std::cerr << "Error: Matrix Market file contains invalid indices." << std::endl;
            exit(EXIT_FAILURE);
        }
        row_indices.push_back(row - 1); // 转换为 0-based 索引
        col_indices.push_back(col - 1);
        values.push_back(value);
    }

    if (row_indices.size() != nnz || col_indices.size() != nnz || values.size() != nnz) {
        std::cerr << "Error: Parsed data does not match expected nnz." << std::endl;
        exit(EXIT_FAILURE);
    }

    file.close();
}

int main() {
    // 1. 读取 Matrix Market 文件
    std::string filename = "C:/Users/ASUS/Downloads/radfr1.tar/radfr1/radfr1.mtx";
    int rows, cols, nnz;
    std::vector<int> row_indices, col_indices;
    std::vector<float> values;

    readMatrixMarket(filename, rows, cols, nnz, row_indices, col_indices, values);

    // 2. 转换为 CSR 格式
    std::vector<int> csrRowPtr(rows + 1, 0);
    for (int i = 0; i < nnz; ++i) {
        ++csrRowPtr[row_indices[i] + 1];
    }
    for (int i = 1; i <= rows; ++i) {
        csrRowPtr[i] += csrRowPtr[i - 1];
    }

    // 3. 在设备上分配内存
    float* d_csrValA, * d_x, * d_y;
    int* d_csrRowPtrA, * d_csrColIndA;
    CUDA_CHECK(hipMalloc((void**)&d_csrValA, nnz * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_csrRowPtrA, (rows + 1) * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_csrColIndA, nnz * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_x, cols * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_y, rows * sizeof(float)));

    CUDA_CHECK(hipMemcpy(d_csrValA, values.data(), nnz * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_csrRowPtrA, csrRowPtr.data(), (rows + 1) * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_csrColIndA, col_indices.data(), nnz * sizeof(int), hipMemcpyHostToDevice));

    // 初始化稠密向量
    std::vector<float> h_x(cols, 1.0f);
    CUDA_CHECK(hipMemcpy(d_x, h_x.data(), cols * sizeof(float), hipMemcpyHostToDevice));

    // 4. 创建 cuSPARSE 描述符
    hipsparseHandle_t handle;
    CUSPARSE_CHECK(hipsparseCreate(&handle));

    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    CUSPARSE_CHECK(hipsparseCreateCsr(&matA, rows, cols, nnz,
        d_csrRowPtrA, d_csrColIndA, d_csrValA,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateDnVec(&vecX, cols, d_x, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateDnVec(&vecY, rows, d_y, HIP_R_32F));

    // 5. 稀疏矩阵-稠密向量乘法，添加计时器
    const float alpha = 1.0f, beta = 0.0f;
    size_t bufferSize = 0;
    void* dBuffer = nullptr;
    CUSPARSE_CHECK(hipsparseSpMV_bufferSize(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY,
        HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
    CUDA_CHECK(hipMalloc(&dBuffer, bufferSize));

    // 使用 CUDA 事件测量时间
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipEventRecord(start, 0));
    CUSPARSE_CHECK(hipsparseSpMV(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, vecX, &beta, vecY,
        HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
    CUDA_CHECK(hipEventRecord(stop, 0));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "Sparse Matrix-Vector Multiplication Time: " << milliseconds << " ms" << std::endl;

    // 6. 清理资源
    CUDA_CHECK(hipFree(d_csrValA));
    CUDA_CHECK(hipFree(d_csrRowPtrA));
    CUDA_CHECK(hipFree(d_csrColIndA));
    CUDA_CHECK(hipFree(d_x));
    CUDA_CHECK(hipFree(d_y));
    CUDA_CHECK(hipFree(dBuffer));
    CUSPARSE_CHECK(hipsparseDestroySpMat(matA));
    CUSPARSE_CHECK(hipsparseDestroyDnVec(vecX));
    CUSPARSE_CHECK(hipsparseDestroyDnVec(vecY));
    CUSPARSE_CHECK(hipsparseDestroy(handle));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}
